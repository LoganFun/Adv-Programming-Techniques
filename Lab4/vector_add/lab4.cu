/*
Author: Zilong Fan
Class:ECE 6122
Last Date Modified: 20231103

Description:
Random 2D Walker
*/

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <ctime>
#include <hiprand/hiprand_kernel.h>
#include <iostream>

#define BLOCK_SIZE 256

inline hipError_t checkCuda(hipError_t result)
{
#if defined(DEBUG) || defined(_DEBUG)
  if (result != cudaSuccess) 
  {
    fprintf(stderr, "CUDA Runtime Error: %s\n", cudaGetErrorString(result));
    assert(result == cudaSuccess);
  }
#endif
  return result;
}


__global__ void randomWalker(float* positions, int numSteps, int numWalkers, 
    unsigned int seed) {
    //Find the id 
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    //Random Initialize
    hiprandState state;
    hiprand_init(seed, idx, 0, &state);

    if (idx < numWalkers) {
        int x = 0;
        int y = 0;

        for (int i = 0; i < numSteps; i++) {

            int direction = (int)(hiprand_uniform(&state) * 4);
            switch (direction) {
                case 3: x++; break; // west
                case 2: x--; break; // east
                case 1: y++; break; // south
                case 0: y--; break; // north
            }
        }

        // Store the final position.
        positions[idx * 2] = x;
        positions[idx * 2 + 1] = y;
        
    }
}

float average(float* positions, int numWalkers) {
    float totalDistance = 0.0;

    for (int i = 0; i < numWalkers; i++) {
        //take the value
        float y = positions[i * 2 + 1];
        float x = positions[i * 2];
        //square x^2 y^2
        float distance = sqrtf(y * y + x * x  );
        totalDistance = totalDistance + distance;
    }

    return totalDistance / numWalkers;
}

void func_CudaMalloc(float* d_positions,float* h_positions, int numSteps, int numWalkers,float& avgDist
    , int numBlocks)
{
    hipMalloc((void**)&d_positions, numWalkers * 2 * sizeof(float));

    // Launch the randomWalk kernel
    
    randomWalker<<<numBlocks, BLOCK_SIZE>>>(d_positions, numSteps, numWalkers, time(NULL));
    checkCuda(hipGetLastError());
    hipDeviceSynchronize();  // Wait for the GPU to finish

    // Copy the positions back to host memory
    hipMemcpy(h_positions, d_positions, numWalkers * 2 * sizeof(float), hipMemcpyDeviceToHost);

    //calculate Distance
    avgDist = average(h_positions, numWalkers);
    
    // Free the memory
    hipFree(d_positions);
    delete[] h_positions;
}

void func_CudaMallocHost(float* d_positions,float* h_positions, int numSteps, int numWalkers,float& avgDist, int numBlocks)
{
    checkCuda( hipHostMalloc((void**)&h_positions, numWalkers * 2 * sizeof(float), hipHostMallocDefault) );
    checkCuda( hipMalloc((void**)&d_positions, numWalkers * 2 * sizeof(float)) );

    randomWalker<<<numBlocks, BLOCK_SIZE>>>(d_positions, numSteps, numWalkers, time(NULL));
    checkCuda(hipGetLastError());
    hipDeviceSynchronize();

    hipMemcpy(h_positions, d_positions, numWalkers * 2 * sizeof(float), hipMemcpyDeviceToHost);
    avgDist = average(h_positions, numWalkers);
    hipFree(d_positions);
    hipHostFree(h_positions);
}

void func_CudaMallocManaged(float* h_positions, int numSteps, int numWalkers,float& avgDist, int numBlocks)
{
    checkCuda( hipMallocManaged(&h_positions, numWalkers * 2 * sizeof(float)) );
    randomWalker<<<numBlocks, BLOCK_SIZE>>>(h_positions, numSteps, numWalkers, time(NULL));
    checkCuda(hipGetLastError());
    hipDeviceSynchronize();
    avgDist = average(h_positions, numWalkers);
    hipFree(h_positions);
}


int main(int argc, char* argv[]) {
    float Time;
    // events for timing
    hipEvent_t startEvent, stopEvent; 
    checkCuda( hipEventCreate(&startEvent) );
    checkCuda( hipEventCreate(&stopEvent) );    

    int numWalkers = atoi(argv[2]);
    int numSteps = atoi(argv[4]);
    int numBlocks = (numWalkers + BLOCK_SIZE - 1) / BLOCK_SIZE;

    float* d_positions;
    float* h_positions = new float[numWalkers * 2];
    float avgDist;

    // 1. Normal CUDA Memory Allocation
    // Start Initial

    func_CudaMallocManaged(h_positions,numSteps, numWalkers,avgDist,numBlocks);

    hipEventRecord(startEvent, 0);

    func_CudaMalloc(d_positions,h_positions,numSteps,numWalkers,avgDist,numBlocks);

    //Stop Initial
    hipEventRecord(stopEvent, 0);
    hipEventSynchronize(stopEvent);
    hipEventElapsedTime(&Time, startEvent, stopEvent);

    // Calculate the average distance on the host
    
    std::cout << "Normal CUDA memory Allocation: " << std::endl;
    std::cout << "  Time to calculate(ms): " << Time*1000 << std::endl;
    std::cout << "  Average distance from origin: " << avgDist << std::endl;     

    // 2. Pinned Memory
    hipEventRecord(startEvent, 0);
    func_CudaMallocHost(d_positions,h_positions,numSteps,numWalkers,avgDist,numBlocks);

    hipEventRecord(stopEvent, 0);
    hipEventSynchronize(stopEvent);
    hipEventElapsedTime(&Time, startEvent, stopEvent);
    
    std::cout << "Pinned Memory Allocation: " << std::endl;
    std::cout << "  Time to calculate(ms): " << Time*1000 << std::endl;
    std::cout << "  Average distance from origin: " << avgDist << std::endl;  

    // 3. Unified Memory
    hipEventRecord(startEvent, 0);
    
    func_CudaMallocManaged(h_positions,numSteps, numWalkers,avgDist,numBlocks);

    hipEventRecord(stopEvent, 0);
    hipEventSynchronize(stopEvent);
    hipEventElapsedTime(&Time, startEvent, stopEvent);
    
    std::cout << "Unified Memory Allocation: " << std::endl;
    std::cout << "  Time to calculate(ms): " << Time*1000 << std::endl;
    std::cout << "  Average distance from origin: " << avgDist << std::endl;  
    
    // Free unified memory
    //cudaFree(h_positions);

    return 0;
}

